#include "my_lib_kernel.h"
#include "stdio.h"

float get_cuda(float* ans, int idx){
    float t=0;
    hipMemcpy(&t, ans+idx, sizeof(float), hipMemcpyDeviceToHost);
    return t;
}

void set_cuda(float* ans, int idx, float t){
    hipMemcpy(ans+idx, &t, sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}